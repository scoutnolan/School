#include "hip/hip_runtime.h"
#include <reduction.h>
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/complex.h>
#include <thrust/execution_policy.h>
#include <thrust/transform.h>
#include <hipcub/hipcub.hpp>

#define COARSE_FACTOR 2
#define BLOCK_DIM 1024

// 1. A sequential CUDA version where one CUDA thread loops over all entries.
__global__ void sequential_reduction_kernel(int* array, int* sum, int elements){
  float tempsum = 0.0f;
  if(threadIdx.x == 0){
      for(int i = 0; i < elements; i++)
          tempsum += array[i];
  }  
  *sum = tempsum;
}

// 2. A parallel CUDA version using atomics (slide 4 of Chapter 10's slides)
__global__ void atomic_reduction_kernel(int* array, int* sum, int elements){
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < elements)
      atomicAdd(sum, array[i]);
}

// 3. A segmented reduction (slide 9 of Chapter 10)
__global__ void segmented_reduction_kernel(int* array, int* sum, int elements){
  unsigned int segment = 2*blockDim.x*blockIdx.x;
  unsigned int i = segment + 2*threadIdx.x;
  for(unsigned int stride = 1; stride <= blockDim.x; stride *=2){
      if(threadIdx.x % stride == 0){
          array[i] += array[i + stride];
      }
      __syncthreads();
  }
  if(threadIdx.x == 0)
    atomicAdd(sum, array[i]);
}

// 4.  A segmented reduction with coalescing and minimal divergence (slide 13 of Chapter 10)
__global__ void coalescing_reduction_kernel(int* array, int* sum, int elements){
  unsigned int segment = 2*blockDim.x*blockIdx.x;
  unsigned int i = segment + threadIdx.x;
  for(unsigned int stride = blockDim.x; stride > 0; stride /=2){
      if(threadIdx.x < stride){
          array[i] += array[i + stride];
      }
      __syncthreads();
  }    
  if(threadIdx.x == 0){
    atomicAdd(sum, array[i]);
  }
}

// 5.  A shared memory segmented reduction (slide 16 of Chapter 10)
__global__ void sharedmem_reduction_kernel(int* array, int* sum, int elements){
  unsigned int segment = 2* blockDim.x * blockIdx.x;
  unsigned int i = segment + threadIdx.x;

  // Load data to shared memory
  __shared__ float array_s[BLOCK_DIM];
  array_s[threadIdx.x] = array[i] + array[i + BLOCK_DIM];
  __syncthreads();

  // Reduction tree in shared memory
  for(unsigned int stride = BLOCK_DIM / 2; stride > 0; stride /= 2)
  {
      if(threadIdx.x < stride){
          array_s[threadIdx.x] += array_s[threadIdx.x + stride];
      }
      __syncthreads();
  }
  if(threadIdx.x == 0)
    atomicAdd(sum, array_s[0]);
}

// 6. A coarsened shared memory segmented reduction (slide 17 of Chapter 10)
__global__ void coarsened_reduction_kernel(int* array, int* sum, int elements){
  unsigned int segment = COARSE_FACTOR * 2 * blockDim.x * blockIdx.x;
  unsigned int i = segment + threadIdx.x;

  // Load data into shared memory
  __shared__ float array_s[BLOCK_DIM];
  float threadSum = 0.0f;
  for(unsigned int c = 0; c < COARSE_FACTOR * 2; ++c){
      threadSum += array[i + c * BLOCK_DIM];
  }
  array_s[threadIdx.x] = threadSum;
  __syncthreads();

  // Reduction tree in shared memory
  for(unsigned int stride = BLOCK_DIM/2; stride > 0; stride /=2){
      if(threadIdx.x < stride){
          array_s[threadIdx.x] += array_s[threadIdx.x + stride];
      }
      __syncthreads();
  }
  if(threadIdx.x == 0)
    atomicAdd(sum, array_s[0]);
}

// 1 
void sequential(int* array, int* sum, int elements){
  int TB_SIZE = BLOCK_DIM;
  int GRID_SIZE = (elements/2 + TB_SIZE - 1) / TB_SIZE;
  sequential_reduction_kernel<<<GRID_SIZE, TB_SIZE>>> (array, sum, elements);
}

// 2
void atomicf(int* array, int* sum, int elements){
  int MIN; int BLOCK;
  checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&MIN, &BLOCK, atomic_reduction_kernel, 0, 0));
  int GRID_SIZE = (elements + BLOCK - 1)/ BLOCK; 
  atomic_reduction_kernel<<<GRID_SIZE, BLOCK>>> (array, sum, elements);
}

// 3
void segmented(int* array, int* sum, int elements){
  int MIN; int BLOCK;
  checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&MIN, &BLOCK, segmented_reduction_kernel, 0, 0));
  int GRID_SIZE = (elements/2 + BLOCK - 1)/ BLOCK; 
  segmented_reduction_kernel<<<GRID_SIZE, BLOCK>>> (array, sum, elements);
}

// 4
void coalescing(int* array, int* sum, int elements){
  int MIN; int BLOCK;
  checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&MIN, &BLOCK, coalescing_reduction_kernel, 0, 0));
  int GRID_SIZE = (elements/2 + BLOCK - 1)/ BLOCK; 
  coalescing_reduction_kernel<<<GRID_SIZE, BLOCK>>> (array, sum, elements);
}

// 5
void sharedmem(int* array, int* sum, int elements){
  int TB_SIZE = BLOCK_DIM;
  int GRID_SIZE = (elements/2 + TB_SIZE - 1) / TB_SIZE;
  sharedmem_reduction_kernel<<<GRID_SIZE, TB_SIZE>>> (array, sum, elements);
}

// 6
void coarsened(int* array, int* sum, int elements){
  int TB_SIZE = BLOCK_DIM;
  int GRID_SIZE = ((elements/2 + TB_SIZE - 1) / TB_SIZE)/COARSE_FACTOR;
  coarsened_reduction_kernel<<<GRID_SIZE, TB_SIZE>>> (array, sum, elements);
}

void CUBCuda(int* array, int* sum, int elements){
  void     *d_temp_storage = NULL;    // CUB temp allocations
  size_t   temp_storage_bytes = 0;
  hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, array, sum, elements); // Storage requirements
  hipMalloc(&d_temp_storage, temp_storage_bytes); // Allocate temporary storage
  hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, array, sum, elements);
}
  
void ThrustCUDA(int* array, int* sum, int elements){
  thrust::device_vector<int> thrustVec(array, array + elements);
  *sum = thrust::reduce(thrustVec.begin(), thrustVec.end());
}