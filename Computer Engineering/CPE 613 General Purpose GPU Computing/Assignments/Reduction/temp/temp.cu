#include "hip/hip_runtime.h"
#include <reduction.h>
#include <hip/hip_runtime_api.h>

#include <thrust/device_vector.h>
#include <hipcub/hipcub.hpp>

#define CUB_IGNORE_DEPRECATED_CPP_DIALECT  CUB_COMPILER_DEPRECATION_SOFT(C++14, C++11)

//SINGLE CUDA THREAD
__global__ void singleCudaThreadKernel(
    int * vector,
    int * res,
    int   n
){

    if(threadIdx.x == 0){
        for(int i = 0; i < n; i++){
            *res += vector[i];
        }
    }
}

void singleCudaThread(
    int   *   vector,
    int   *   res,
    int       n
) {

    singleCudaThreadKernel <<< 1, 1>>>(vector, res, n);

    checkCudaErrors(hipGetLastError());

}

// ATOMIC VARIABLES
__global__ void atomicVarKernel(
    int * vector,
    int * res,
    int n
){
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < n) atomicAdd(res, vector[i]);
}

void atomicVar(
    int   *   vector,
    int   *   res,
    int       n
) {
    int blockSize = BLOCK_DIM;
    int gridSize = ceil((n + blockSize - 1)/blockSize);

    atomicVarKernel <<< gridSize, blockSize >>>(vector, res, n);

    checkCudaErrors(hipGetLastError());
}

//SEGMENTED
__global__ void segmentedKernel(
    int *   vector,
    int *   res
) {
    unsigned int segment = 2 * blockDim.x * blockIdx.x;
    unsigned int i = segment + 2 * threadIdx.x;

    for(unsigned int stride = 1; stride <= blockDim.x; stride *= 2){
        if(threadIdx.x % stride == 0)
            vector[i] += vector[i + stride]; 
        __syncthreads();
    }


    if(threadIdx.x == 0)
        atomicAdd(res, vector[i]);
}

void segmented(
    int *   vector,
    int *   res,
    int     n
) {
    int blockSize, min;
    checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
        &min, 
        &blockSize, 
        segmentedKernel, 0, 0)
    );

    int gridSize = (n/2 + blockSize-1)/blockSize;

    segmentedKernel <<< gridSize, blockSize >>> (vector, res);

    checkCudaErrors(hipGetLastError());

}

//COALESCING
__global__ void coalescedKernel(
    int *   vector,
    int *   res
) {
    unsigned int segment = 2 * blockDim.x * blockIdx.x;
    unsigned int i = segment + threadIdx.x;

    for(unsigned int stride = blockDim.x; stride > 0; stride /= 2){
        if(threadIdx.x < stride)
            vector[i] += vector[i + stride];
        __syncthreads();
    }

  if(threadIdx.x == 0)
    atomicAdd(res, vector[i]);
}

void coalesced(
    int * vector,
    int * res,
    int   n
) {
    int blockSize, min;
    checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
        &min, 
        &blockSize, 
        coalescedKernel, 0, 0)
    );

    int gridSize = (n/2 + blockSize-1)/blockSize;

    coalescedKernel <<< gridSize, blockSize >>> (vector, res);

    checkCudaErrors(hipGetLastError());
}

//SHARED SEGMENTED
__global__ void sharedSegmentedKernel(
    int *   vector,
    int *   res
) {
    unsigned int segment = 2 * blockDim.x * blockIdx.x;
    unsigned int i = segment + threadIdx.x;

    __shared__ int vectorShared[BLOCK_DIM];
    vectorShared[threadIdx.x] = vector[i] + vector[i + BLOCK_DIM];
    __syncthreads();

    for(unsigned int stride = BLOCK_DIM/2; stride > 0; stride /= 2) {
        if(threadIdx.x < stride)
            vectorShared[threadIdx.x] += vectorShared[threadIdx.x + stride];

        __syncthreads();
    }

    if(threadIdx.x == 0)
        atomicAdd(res, vectorShared[0]);
}

void sharedSegmented(
    int *   vector,
    int *   res,
    int     n
) {
    int blockSize = BLOCK_DIM;
    int gridSize = (n/2 + blockSize-1)/blockSize;

    sharedSegmentedKernel <<< gridSize, blockSize >>> (vector, res);

    checkCudaErrors(hipGetLastError());
}

//COARSENED SHARED SEGMENTED
__global__ void sharedSegmentedCoarsenedKernel(
    int *   vector,
    int *   res
) {
    unsigned int segment = COARSE_FACTOR * 2 * blockDim.x * blockIdx.x;
    unsigned int i = segment + threadIdx.x;

    __shared__ int vectorShared[BLOCK_DIM];
    int threadSum = 0;

    for(unsigned int c = 0; c < COARSE_FACTOR * 2; ++c){
        threadSum += vector[i + c*BLOCK_DIM];
    }

    vectorShared[threadIdx.x] = threadSum;
    __syncthreads();

    for(unsigned int stride = BLOCK_DIM/2; stride > 0; stride /= 2) {
        if(threadIdx.x < stride)
            vectorShared[threadIdx.x] += vectorShared[threadIdx.x + stride];

        __syncthreads();
    }
    

    atomicAdd(res, vectorShared[i]);

}

void sharedSegmentedCoarsened(
    int *   vector,
    int *   res,
    int     n
) {
    int blockSize = BLOCK_DIM;
    int gridSize = (n + blockSize - 1)/blockSize + 1;

    sharedSegmentedCoarsenedKernel <<< gridSize, blockSize >>> (vector, res);

    checkCudaErrors(hipGetLastError());
}

//TESTING AGAINST THRUST AND CUB
void verify(
    char v, 
    int *vector,
    int *res, 
    int n
) {
    
    if(v == 't'){
        thrust::device_vector<int> thrustVec(vector, vector + n);
        *res =  thrust::reduce(thrustVec.begin(), thrustVec.end());
    }
    else if(v == 'c'){
        
        void *tempStore = NULL;
        size_t storeBytes = 0;

        hipcub::DeviceReduce::Sum(tempStore, storeBytes, vector, res, n);
        
        hipMalloc(&tempStore, storeBytes);

        hipcub::DeviceReduce::Sum(tempStore, storeBytes, vector, res, n);

    }
}