#include "hip/hip_runtime.h"
#include <matmul.h>
#include <hip/hip_runtime_api.h>
#define TILE_WIDTH 16
#define COARSE_FACTOR 4

__global__ void MatrixMulKernel(float* A, float* B, float* C, int Size)
{
  int Row = blockIdx.y * blockDim.y + threadIdx.y;
  int Col = blockIdx.x * blockDim.x + threadIdx.x;
  if((Row < Size) && (Col < Size)){
    float Cvalue = 0;
    for(int k = 0; k < Size; ++k){
      Cvalue += A[Row * Size + k] * B[k * Size + Col];
    }
    C[Row * Size + Col] = Cvalue;
  }
}

__global__ void MatrixMulKernelTile(float * A, float * B, float * C, int Size) { 
  __shared__ float Ads[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Bds[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
  int row = by * TILE_WIDTH + ty;
  int col = bx * TILE_WIDTH + tx;
  float CValue = 0.0f;
  for(unsigned int ph = 0; ph < ceil(Size/(float)TILE_WIDTH); ++ph){
      if((row < Size) && (ph * TILE_WIDTH + tx) < Size)
        Ads[ty][tx] = A[row * Size + ph * TILE_WIDTH + tx];
      if((ph * TILE_WIDTH + ty) < Size && col < Size)
        Bds[ty][tx] = B[(ph * TILE_WIDTH + ty) * Size + col];
      __syncthreads();
      for(unsigned int i = 0; i < TILE_WIDTH; ++i)
          CValue += Ads[ty][i] * Bds[i][tx];
      __syncthreads();
  }
  if(row < Size && col < Size)
    C[row * Size + col] = CValue;
}

__global__ void MatrixMulKernelCoarse(float * A, float * B, float * C, int Size) { 
  __shared__ float Ads[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Bds[TILE_WIDTH][TILE_WIDTH];
  unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int colStart = blockIdx.x * blockDim.x + threadIdx.x;
  float sum[COARSE_FACTOR];
  for(unsigned int c = 0; c < COARSE_FACTOR; ++c){
    sum[c] = 0.0f;
  }    

  for(unsigned int tile = 0; tile < Size / TILE_WIDTH; ++tile)
  {
    if((row < Size) && (tile * TILE_WIDTH + threadIdx.x) < Size)
      Ads[threadIdx.y][threadIdx.x] = A[row * Size + tile * TILE_WIDTH + threadIdx.x];
    for(unsigned int c = 0; c < COARSE_FACTOR; ++c){
      unsigned int col = colStart + c * TILE_WIDTH;
      if((tile * TILE_WIDTH + threadIdx.y) < Size && col < Size)
        Bds[threadIdx.y][threadIdx.x] = B[(tile*TILE_WIDTH + threadIdx.y)*Size + col];
      __syncthreads();
      for(unsigned int i = 0; i < TILE_WIDTH; ++i){
        sum[c] += Ads[threadIdx.y][i] * Bds[i][threadIdx.x];
      }
      __syncthreads();
    }
    for(unsigned int c = 0; c < COARSE_FACTOR; ++c){
      unsigned int col = colStart + c*TILE_WIDTH;
      if(row < Size && col < Size)
        C[row*Size + col] = sum[c];
    }
  }
}

void MatrixMul(float* A, float* B, float* C, int Size)
{
  int blockSize;
  int minGridSize;
  checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, MatrixMulKernel, 0, 0));

  dim3 numThreadsPerBlock(16, 16);
  dim3 numBlocks((Size + numThreadsPerBlock.x - 1)/numThreadsPerBlock.x,
                  (Size + numThreadsPerBlock.y - 1)/numThreadsPerBlock.y);

  MatrixMulKernel<<<numThreadsPerBlock, numBlocks>>> (A, B, C, Size); 
}

void MatrixMulTile(float* A, float* B, float* C, int Size){
  int blockSize;
  int minGridSize;
  checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, MatrixMulKernelTile, 0, 0));
  int numThread=16; //number of threads per Block axis
  int numBlocks=Size/numThread;
  if(Size%numThread)
      numBlocks++;
  dim3 dimGrid(numBlocks,numBlocks);
  dim3 dimBlock(numThread,numThread);
  MatrixMulKernelTile<<<dimGrid,dimBlock>>> (A, B, C, Size); 
}

void MatrixMulCoarse(float* A, float* B, float* C, int Size)
{
  int blockSize;
  int minGridSize;
  checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, MatrixMulKernelCoarse, 0, 0));  
  int numThread=16; //number of threads per Block axis
  int numBlocks=Size/numThread;
  if(Size%numThread)
      numBlocks++;
  dim3 dimGrid(numBlocks,numBlocks);
  dim3 dimBlock(numThread,numThread);
  MatrixMulKernelCoarse<<<dimGrid,dimBlock>>> (A, B, C, Size); 
}

