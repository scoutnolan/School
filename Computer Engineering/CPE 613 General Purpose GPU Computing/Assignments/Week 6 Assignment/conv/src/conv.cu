#include "hip/hip_runtime.h"
#include <conv.h>
#include <hip/hip_runtime_api.h>

#define FILTER_RADIUS 2
__constant__ float constantFilter[(2*FILTER_RADIUS+1)* (2*FILTER_RADIUS+1)];
#define TILE_WIDTH 4
#define IN_TILE_DIM 16
#define OUT_TILE_DIM ((IN_TILE_DIM) - 2 * (FILTER_RADIUS))

// Number 1 naive approach. Load image and filter from main memory.
__global__ void convolution_2D_basic_kernel(float* N, float *F, float *P, int r, int width, int height){
  int outCol = blockIdx.x * blockDim.x + threadIdx.x;
  int outRow = blockIdx.y * blockDim.y + threadIdx.y;
  float Pvalue = 0.0f;
  if(outCol < width && outRow < height){
    for(int fRow = 0; fRow < 2*r+1; fRow++){
      for(int fCol = 0; fCol < 2*r+1; fCol++){
        int inRow = outRow - r + fRow;
        int inCol = outCol - r + fCol;
        if(inRow >= 0 && inRow < height && inCol >=0 && inCol < width){
          Pvalue += F[fRow * r + fCol] * N[inRow * width + inCol];
        }
      }
    }
    P[outRow * width + outCol] = Pvalue;
  }
}

// Number 2 filter in shared memory approach.
__global__ void convolution_2D_filter_shared_mem_kernel(float* N, float *F, float *P, int r, int width, int height){
  int outCol = blockIdx.x * blockDim.x + threadIdx.x;
  int outRow = blockIdx.y * blockDim.y + threadIdx.y;

  __shared__ float filter[FILTER_RADIUS * FILTER_RADIUS];
  float Pvalue = 0.0f;
  if(outCol < width && outRow < height){
    for(int row = 0; row < 2 * r + 1; row++){
      for(int col = 0; col < 2 * r + 1; col++){
        filter[row * r + col] = F[row * r + col];
      }
    }
    __syncthreads();

    for(int fRow = 0; fRow < 2 * r+1; fRow++){
      for(int fCol = 0; fCol < 2 * r+1; fCol++){
        int inRow = outRow - r + fRow;
        int inCol = outCol - r + fCol;
        if(inRow >= 0 && inRow < height && inCol >=0 && inCol < width){
          Pvalue += filter[fRow * r + fCol] * N[inRow * width + inCol];
        }
      }
    }
    P[outRow * width + outCol] = Pvalue;
  }
}

// Number 3 filter in constant memory.
__global__ void convolution_2D_const_mem_kernel(float* N, float *P, int r, int width, int height){
  int outCol = blockIdx.x * blockDim.x + threadIdx.x;
  int outRow = blockIdx.y * blockDim.y + threadIdx.y;
  float Pvalue = 0.0f;
  if(outCol < width && outRow < height){
    for(int fRow = 0; fRow < 2*r+1; fRow++){
      for(int fCol = 0; fCol < 2*r+1; fCol++){
        int inRow = outRow - r + fRow;
        int inCol = outCol - r + fCol;
        if(inRow >= 0 && inRow < height && inCol >=0 && inCol < width){
          Pvalue += constantFilter[fRow * r + fCol] * N[inRow * width + inCol];
        }
      }
    }
    P[outRow * width + outCol] = Pvalue;
  }
}

// Number 4 - tile input to shared memory, and load filter into shared memory. 
__global__ void convolution_2D_input_shared_filter_const_kernel(float* N, float *F, float *P, int r, int width, int height){

  int outCol = blockIdx.x * OUT_TILE_DIM + threadIdx.x - FILTER_RADIUS;
  int outRow = blockIdx.y * OUT_TILE_DIM + threadIdx.y - FILTER_RADIUS;
  __shared__ float N_s[IN_TILE_DIM][IN_TILE_DIM];
  __shared__ float filter[FILTER_RADIUS * FILTER_RADIUS];
  
  if(outCol < width && outCol >= 0 && outRow >= 0 && outRow < height){
    N_s[threadIdx.y][threadIdx.x] = N[outRow * width + outCol];
  }else{
    N_s[threadIdx.y][threadIdx.x] = 0.0;
  }
  __syncthreads();
  
  if(outCol < width && outCol >= 0 && outRow >= 0 && outRow < height){
    // Pull the filter into shared memory
    for(int row = 0; row < 2*FILTER_RADIUS + 1; row++){
      for(int col = 0; col < 2*FILTER_RADIUS + 1; col++){
        filter[row * FILTER_RADIUS * FILTER_RADIUS + col] = F[row * FILTER_RADIUS * FILTER_RADIUS + col];
      }
    }
    __syncthreads();

    int tCol = threadIdx.x - FILTER_RADIUS;
    int tRow = threadIdx.y - FILTER_RADIUS;
    float Pvalue = 0.0f;

    if(tCol >= 0 && tCol < OUT_TILE_DIM && tRow >= 0 && tRow < OUT_TILE_DIM){
      for(int row = 0; row < 2 * FILTER_RADIUS + 1; row++){
        for(int col = 0; col < 2 * FILTER_RADIUS + 1; col++){
          if(outCol < width && outRow < height)
            Pvalue += filter[row * FILTER_RADIUS + col] * N_s[tRow + row][tCol + col];
        }
      }
      P[outRow * width + outCol] = Pvalue;
    }
  }
}

// Number 5 - Tile input to shared memory, load filter to constant memory.
__global__ void convolution_tiled_2D_const_mem_kernel(float *N, float *P, int r, int width, int height)
{
  int outCol = blockIdx.x * OUT_TILE_DIM + threadIdx.x - FILTER_RADIUS;
  int outRow = blockIdx.y * OUT_TILE_DIM + threadIdx.y - FILTER_RADIUS;
  __shared__ float N_s[IN_TILE_DIM][IN_TILE_DIM];
  
  if(outCol < width && outCol >= 0 && outRow >= 0 && outRow < height){
    N_s[threadIdx.y][threadIdx.x] = N[outRow * width + outCol];
  }else{
    N_s[threadIdx.y][threadIdx.x] = 0.0;
  }
  __syncthreads();


  if(outCol >= 0 && outCol < width && outRow >= 0 && outRow < height){
    int tileCol = threadIdx.x - FILTER_RADIUS;
    int tileRow = threadIdx.y - FILTER_RADIUS;
    float Pvalue = 0.0f;
    if(tileCol >= 0 && tileCol < OUT_TILE_DIM && tileRow >= 0 && tileRow < OUT_TILE_DIM){
      for(int fRow = 0; fRow < 2 * FILTER_RADIUS + 1; fRow++){
        for(int fCol = 0; fCol < 2 * FILTER_RADIUS + 1; fCol++){
            Pvalue += constantFilter[fRow * r + fCol] * N_s[tileRow + fRow][tileCol + fCol];
        }
      }
      P[outRow * width + outCol] = Pvalue;
    }
  }
}

// Number 6 - Tile input to sharead memory where in/out tile sizes are identical and load the filter into constant memory. 
__global__ void convolution_cached_tiled_2D_const_mem_kernel(float *N, float *P, int r, int width, int height)
{
  int col = blockIdx.x * OUT_TILE_DIM + threadIdx.x - FILTER_RADIUS;
  int row = blockIdx.y * OUT_TILE_DIM + threadIdx.y - FILTER_RADIUS;
  
  __shared__ float N_s[IN_TILE_DIM][IN_TILE_DIM];
  if(row < height & col < width){
    N_s[threadIdx.y][threadIdx.x] = N[row * width + col];
  } else {
    N_s[threadIdx.y][threadIdx.x] = 0.0;
  }
  __syncthreads();

  if(col < width && row < height){
    float Pvalue = 0.0f;
    for(int fRow = 0; fRow < 2 * FILTER_RADIUS + 1; fRow++){
      for(int fCol = 0; fCol < 2 * FILTER_RADIUS + 1; fCol++){
        if(threadIdx.x - FILTER_RADIUS + fCol >= 0 &&
           threadIdx.x - FILTER_RADIUS + fCol < TILE_WIDTH && 
           threadIdx.y - FILTER_RADIUS + fRow >= 0 &&
           threadIdx.y - FILTER_RADIUS + fRow < TILE_WIDTH){
            Pvalue += constantFilter[fRow * r + fCol] * N_s[threadIdx.y + fRow][threadIdx.x + fCol];
        }
        else{
          if(row - FILTER_RADIUS + fRow >= 0 &&
             row - FILTER_RADIUS + fRow < height &&
             col - FILTER_RADIUS + fCol >= 0 &&
             col - FILTER_RADIUS + fCol < width){
              Pvalue += constantFilter[fRow * r + fCol] * N[(row - FILTER_RADIUS + fRow) * width + col - FILTER_RADIUS + fCol];
          }
        }
      }
      P[row * width + col] = Pvalue;
    }
  }
}

void Number1(float* N, float *F, float *P, int r, int width, int height)
{
  dim3 blocks(16, 16);
  dim3 grid((width + blocks.x - 1)/blocks.x, (height + blocks.y - 1)/blocks.y);
  convolution_2D_basic_kernel<<<grid, blocks>>> (N, F, P, r, width, height);
}

void Number2(float* N, float *F, float *P, int r, int width, int height){
  dim3 blocks(16, 16);
  dim3 grid((width + blocks.x - 1)/blocks.x, (height + blocks.y - 1)/blocks.y);
  convolution_2D_filter_shared_mem_kernel<<<grid, blocks>>> (N, F, P, r, width, height);
}

void Number3(float* N, float *P, int r, int width, int height){
  dim3 blocks(16, 16);
  dim3 grid((width + blocks.x - 1)/blocks.x, (height + blocks.y - 1)/blocks.y);
  convolution_2D_const_mem_kernel<<<grid, blocks>>> (N, P, r, width, height);
}

void Number4(float* N, float *F, float *P, int r, int width, int height){
  dim3 blocks(16, 16);
  dim3 grid((width + blocks.x - 1)/blocks.x, (height + blocks.y - 1)/blocks.y);
  convolution_2D_input_shared_filter_const_kernel<<<grid, blocks>>> (N, F, P, r, width, height);
}

void Number5(float *N, float *P, int r, int width, int height){
  dim3 blocks(16, 16);
  dim3 grid((width + blocks.x - 1)/blocks.x, (height + blocks.y - 1)/blocks.y);
  convolution_tiled_2D_const_mem_kernel<<<grid, blocks>>> (N, P, r, width, height);
}

void Number6(float *N, float *P, int r, int width, int height){
  dim3 blocks(16, 16);
  dim3 grid((width + blocks.x - 1)/blocks.x, (height + blocks.y - 1)/blocks.y);
  convolution_cached_tiled_2D_const_mem_kernel<<<grid, blocks>>> (N, P, r, width, height);
}