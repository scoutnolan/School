#include "hip/hip_runtime.h"
#include <convolution.h>
#include <hip/hip_runtime_api.h>

#define FILTRAD 5

__constant__ float constFilt[FILTRAD*FILTRAD];

//EXERCISE 1
__global__ void convolution_naive_kernel(
    int         width,
    int         height,
    int         filtRad,
    float   *   grayImageDevice,
    float   *   filtImageDevice,
    float   *   filterDevice
) {
    int outCol = blockIdx.x * blockDim.x + threadIdx.x;
    int outRow = blockIdx.y * blockDim.y + threadIdx.y;

    float pVal = 0.0f;
    if(outCol < width && outRow < height)
        for(int r = 0; r < 2 * filtRad + 1; r++){
            for(int c = 0; c < 2 * filtRad + 1; c++){
                int inRow = outRow - filtRad + r; 
                int inCol = outCol - filtRad + c;

                if(inRow >= 0 && inRow < height && inCol >= 0 && inCol < width){
                    pVal += filterDevice[r * filtRad + c] * grayImageDevice[inRow * width + inCol];
                }
            }
        }

        filtImageDevice[outRow * width + outCol] = pVal;
    }

void convolutionNaive(
    int     width,
    int     height,
    int     filtRad,
    float * grayImageDevice,
    float * filtImageDevice,
    float * filtGaussDevice
) {

    dim3 numBlocks(16,16);
    dim3 gridSize((width + numBlocks.x - 1)/numBlocks.x,
                  (height + numBlocks.y - 1)/numBlocks.y);

    convolution_naive_kernel <<< gridSize, numBlocks >>>(
        width,
        height,
        filtRad,
        grayImageDevice,
        filtImageDevice,
        filtGaussDevice
    );

    checkCudaErrors(hipGetLastError());
}

//EXERCISE 2

__global__ void convolution_shared_kernel(
    int     width,
    int     height,
    int     filtRad,
    float * grayImageDevice,
    float * filtImageDevice,
    float * filterDevice
) {
    
    int outCol = blockIdx.x * blockDim.x + threadIdx.x;
    int outRow = blockIdx.y * blockDim.y + threadIdx.y;

    __shared__ float filter[FILTRAD*FILTRAD];

    float pVal = 0.0f;

    if(outCol < width && outRow < height){
        //load filter
        for(int r = 0; r < 2*filtRad + 1; r++){
            for(int c = 0; c < 2*filtRad + 1; c++){
                filter[r * filtRad*filtRad + c] = filterDevice[r * filtRad*filtRad + c];
            }
        }

        __syncthreads();

        for(int r = 0; r < 2 * filtRad + 1; r++){
            for(int c = 0; c < 2 * filtRad + 1; c++){
                int inRow = outRow - filtRad + r; 
                int inCol = outCol - filtRad + c;

                if(inRow >= 0 && inRow < height && inCol >= 0 && inCol < width){
                    pVal += filter[r * filtRad + c] * grayImageDevice[inRow * width + inCol];
                }
            }
        }

        filtImageDevice[outRow * width + outCol] = pVal;
    }
}


void convolutionShared(
    int     width,
    int     height,
    int     filtRad,
    float * grayImageDevice,
    float * filtImageDevice,
    float * filtGaussDevice
) {

    dim3 numBlocks(16,16);
    dim3 gridSize((width + numBlocks.x - 1)/numBlocks.x,
                  (height + numBlocks.y - 1)/numBlocks.y);

    convolution_shared_kernel <<< gridSize, numBlocks >>>(
        width,
        height,
        filtRad,
        grayImageDevice,
        filtImageDevice,
        filtGaussDevice
    );

    checkCudaErrors(hipGetLastError());
}

//EXERCISE 3
__global__ void convolution_const_kernel(
    int         width,
    int         height,
    int         filtRad,
    float   *   grayImageDevice,
    float   *   filtImageDevice
) {
    int outCol = blockIdx.x * blockDim.x + threadIdx.x;
    int outRow = blockIdx.y * blockDim.y + threadIdx.y;

    float pVal = 0.0f;
    if(outCol < width && outRow < height)
        for(int r = 0; r < 2 * filtRad + 1; r++){
            for(int c = 0; c < 2 * filtRad + 1; c++){
                int inRow = outRow - filtRad + r; 
                int inCol = outCol - filtRad + c;

                if(inRow >= 0 && inRow < height && inCol >= 0 && inCol < width){
                    pVal += constFilt[r * filtRad + c] * grayImageDevice[inRow * width + inCol];
                }
            }
        }

        filtImageDevice[outRow * width + outCol] = pVal;
    }

void convolutionConst(
    int     width,
    int     height,
    int     filtRad,
    float * grayImageDevice,
    float * filtImageDevice
) {

    dim3 numBlocks(16,16);
    dim3 gridSize((width + numBlocks.x - 1)/numBlocks.x,
                  (height + numBlocks.y - 1)/numBlocks.y);

    convolution_const_kernel <<< gridSize, numBlocks >>>(
        width,
        height,
        filtRad,
        grayImageDevice,
        filtImageDevice
    );

    checkCudaErrors(hipGetLastError());
}

