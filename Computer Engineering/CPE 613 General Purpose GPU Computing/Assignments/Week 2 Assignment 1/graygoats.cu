/*
    Nolan Anderson
    CPE 613 Week 2 Assignment 2
    This program takes in an image that has been converted to a text file.
    This text file has the RGB values. I converted the file in matlab using the following code:
    rgb = imread('goats.jpeg');
    writematrix(rgb,'goats.txt', 'delimiter', ' ');
    ^^^^^ MAKE SURE TO PUT THE DEMINSIONS ON THE FIRST ROW OF THE TEXT FILE ^^^^^
    ----------------------------------------------------
    After this program, you should be able to run the following in matlab and see the grayscale image:
    filename = 'graygoats.txt'
    grayImage = uint8(importdata(filename));
    imwrite(grayImage, 'myimage.png');
    ----------------------------------------------------
    To compile, first 'load module cuda'
    Then: 'nvcc graygoats.cu -o goats'

    Next, 'run_gpu goats' and use all default values. 
    This should result in a grayscale data of your original image.
*/


#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
using namespace std;

// RGB to gray kernel. Pretty much just what was in the slides.
__global__ void rgb2gray_kernel(unsigned char* r, unsigned char* g, unsigned char* b, unsigned char* gray, int matrixHeight, int matrixWidth){
    unsigned int row = blockIdx.y*blockDim.y + threadIdx.y;     // Row and column values
    unsigned int col = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int index = row*matrixWidth + col;
    if(row < matrixHeight && col < matrixWidth)                // Make sure we don't go out of bounds on the row & column
        gray[index] = r[index] *3/10 + g[index]*6/10 + b[index]*1/10; // General calculation for RGB -> gray values.
}

int main(){
    // Allocate matrices, first using matrixWidth and matrixHeight.
    // The data is coming from goats.txt, or any image whose data is in 
    // text format.
    ifstream inFile;                        // input file
    int matrixWidth, matrixHeight;          // matrixWidth and matrixHeight of the matrix.
    inFile.open("goats.txt");            // open the input file
    inFile >> matrixWidth >> matrixHeight;  // pull the matrix deminsions out of the file. (on the first line)
    const int matrixSize = matrixWidth * matrixHeight; // Obtain the matrix size. 
    unsigned char  *r, *g, *b, *gray;       // Create the pointers to hold the rgb data and the grayscale data. 

    // Allocate a matrix for each color. 
    unsigned char grayMatrix[matrixSize], rMatrix[matrixSize], gMatrix[matrixSize], bMatrix[matrixSize];
    
    // Populate each matrix with the input file values
    int temp;
    for(int r = 0; r < matrixSize; r++){    // Loop through number of values in matrix
        inFile >> temp;
        rMatrix[r] = temp;                // Assign to each matrix. 
        inFile >> temp;
        gMatrix[r] = temp;        
        inFile >> temp;
        bMatrix[r] = temp;        
    }
    
    // Allocate space on the device for rgb and gray matrices. 
    // The size is the size of an unsigned char, multiplied by the size of the matrix. 
    hipMalloc((void**)&r, matrixSize*sizeof(unsigned char));
    hipMalloc((void**)&g, matrixSize*sizeof(unsigned char));
    hipMalloc((void**)&b, matrixSize*sizeof(unsigned char));
    hipMalloc((void**)&gray, matrixSize*sizeof(unsigned char));

    // Copy the rgb values to the device. Make sure to leave out the gray value. This one will only be
    // copied back to the host. 
    hipMemcpy(r, &rMatrix, matrixSize * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(g, &gMatrix, matrixSize * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(b, &bMatrix, matrixSize * sizeof(unsigned char), hipMemcpyHostToDevice);
    
    // Create the dim3. 1024x1024 threads per block. Just using the standard number of blocks from
    // the slides. 
    dim3 numThreadsPerBlock(1024,1024);
    dim3 numBlocks((matrixWidth + numThreadsPerBlock.x - 1)/numThreadsPerBlock.x,
                    (matrixHeight + numThreadsPerBlock.y - 1)/numThreadsPerBlock.y);

    // Call the kernel, passing the appropriate values in. We need to add the dim3 in the <<< >>> so the
    // warps / kernel / SMs know how to allocate their data. Gray is the output and matrixheight/width are
    // there to support the calculations and indexing.
    rgb2gray_kernel <<< numThreadsPerBlock, numBlocks >>> (r, g, b, gray, matrixHeight, matrixWidth);
    
    // Copy the data back to host from the device. 
    hipMemcpy(grayMatrix, gray, matrixSize*sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Write the data to an outfile. In this case we're going back to a text file. 
    ofstream outFile;               // osftream variable
    outFile.open("graygoats.txt");  // open the file 
    for(int row = 0; row < matrixHeight; row++){        // Loop through the rows
        for(int col = 0; col < matrixWidth; col++)      // Loop through the columns
            outFile << +grayMatrix[row*matrixWidth + col] << " ";   // go to index to get current column in row. 
        outFile << endl;    // Move to a new line (we're done with the previous row)
    }

    // Close the files and exit program.     
    inFile.close();
    outFile.close();
    return 0;
}